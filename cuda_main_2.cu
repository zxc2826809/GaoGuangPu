#include <stdio.h>
#include <stdlib.h>
#include <time.h>

//CUDA RunTime API
#include <hip/hip_runtime.h>

#include "hip/device_functions.h"

#define THREAD_NUM 256

#define raw_row 512*680
#define raw_column 224

#define MATRIX_SIZE raw_row*raw_column

const int blocks_num = 16;


//// __global__ 函数 并行计算矩阵乘法
//__global__ static void matMultCUDA(Imagdata* imagdata, CovData* imagecov, cBYTE* datatemp, clock_t* time)
//{
//
//	//表示目前的 thread 是第几个 thread（由 0 开始计算）
//	const int tid = threadIdx.x;
//
//	//表示目前的 thread 属于第几个 block（由 0 开始计算）
//	const int bid = blockIdx.x;
//
//	//// 只适用方阵
//	////从 bid 和 tid 计算出这个 thread 应该计算的 row 和 column
//	//const int idx = bid * THREAD_NUM + tid;
//	//const int row = idx / n;
//	//const int column = idx % n;
//
//	//记录运算开始的时间
//
//	//只在 thread 0（即 threadIdx.x = 0 的时候）进行记录，每个 block 都会记录开始时间及结束时间
//	if (tid == 0) time[bid] = clock();
//	if (tid == 0)
//		printf("Hello thread %d, f=%f\n", tid);
//
//	/*
//	计算协方差过程开始
//	*/
//
//	long count;
//	long count2;
//
//	imagecov->row = imagdata->Dim;
//	imagecov->column = imagdata->Dim;
//
//	for (int i = 0; i < imagdata->Dim; ++i) {
//		count = 0;
//		int j = 0;
//		for (j = 0; j < imagdata->imgSize; ++j) {
//			count += imagdata->data[i * imagdata->imgSize + j];
//		}
//		int mean = count / imagdata->imgSize;
//		for (j = 0; j < imagdata->imgSize; ++j) {
//			datatemp[i * imagdata->imgSize + j] = imagdata->data[i * imagdata->imgSize + j] - mean;
//		}
//	}
//	//cout << "(imgcompute.cpp)像素均值差计算完成\n" << endl;
//	printf("(imgcompute.cpp)像素均值差计算完成...\n");
//	//-----------------------------------------------------------------------------------------------------------
//	//此部分运行时间为45秒。并发处理后为21秒。将循环次减半，时间缩短为12s
//
//	for (int i = 0; i < imagdata->Dim; ++i)
//	{
//		int j;
//		for (j = i; j < imagdata->Dim; ++j)
//		{
//			count2 = 0;
//			int k;
//			//for (k = 0; k < imagdata->imgSize; ++k) {
//
//			//	count2 += datatemp[i * imagdata->imgSize + k] * datatemp[j * imagdata->imgSize + k];
//			//}
//			for (k = 0; k < imagdata->imgSize; ++k) {
//
//				count2 += datatemp[i * imagdata->imgSize + k] * datatemp[j * imagdata->imgSize + k];
//			}
//			imagecov->data[i * (imagecov->row) + j] = count2 / (imagdata->imgSize - 1);
//			if (i != j) {
//				imagecov->data[j * (imagecov->row) + i] = count2 / (imagdata->imgSize - 1);
//			}
//		}
//	}
//
//	delete[]datatemp;
//	printf("(imgcompute.cpp)协方差计算完成\n");
//
//	/*
//	计算协方差过程结束
//	*/
//
//
//	//计算时间,记录结果，只在 thread 0（即 threadIdx.x = 0 的时候）进行，每个 block 都会记录开始时间及结束时间
//	if (tid == 0)
//	{
//		time[bid + blocks_num] = clock();
//	}
//
//}
//
//
//__global__ void cudatest() {
//	printf("success!");
//}
//
//
//extern "C" void cuda_juzhen(RawImag * myraw, Imagdata * imagdata, CovData * imagecov)
//{
//	clock_t* time;
//
//	printf("1\n");
//
//	//CPU和GPU分配内存 同一个结构体要分开声明（实例化）
//	//CPU分配内存
//	imagecov->data = new covBYTE[sizeof(covBYTE) * imagdata->Dim * imagdata->Dim];
//	cBYTE* datatemp = new cBYTE[sizeof(cBYTE) * imagdata->rawSize]; // rawSize 是512x680x224,8000W
//
//	printf("2\n");
//
//
//	//cudaMalloc 取得一块显卡内存 
//	//结构体分配内存要分配结构体本身，以及结构体中每一个变量
//
//
//	Imagdata* cuda_imagdata;
//	CovData* cuda_imagecov;
//	cBYTE* cuda_datatemp;
//
//	cuda_imagdata = new Imagdata[sizeof(Imagdata)]; // 申请结构体数组（指针）的内存
//	cuda_imagecov = new CovData[sizeof(CovData)];
//	cuda_datatemp = new cBYTE[sizeof(cBYTE)];
//
//
//	//memset(&cuda_imagdata, 0, sizeof(Imagdata));
//	//memset(&cuda_imagecov, 0, sizeof(CovData));
//	//memset(&cuda_datatemp, 0, sizeof(cBYTE));
//
//
//	cuda_imagdata->Col = imagdata->Col;
//	cuda_imagdata->Row = imagdata->Row;
//	cuda_imagdata->Dim = imagdata->Dim;
//	cuda_imagdata->rawSize = imagdata->rawSize;
//	cuda_imagdata->imgSize = imagdata->imgSize;
//
//	cuda_imagecov->row = imagdata->Dim;
//	cuda_imagecov->column = imagdata->Dim;
//
//
//	cuda_imagdata->data = new covBYTE[sizeof(covBYTE) * imagdata->Dim * imagdata->Dim];;
//	cuda_datatemp = new cBYTE[sizeof(cBYTE) * imagdata->rawSize]; // rawSize 是512x680x224,8000W;
//
//	//cudaMalloc((void**)&cuda_imagdata->Row, sizeof(int));
//	//cudaMalloc((void**)&cuda_imagdata->Col, sizeof(int));
//	//cudaMalloc((void**)&cuda_imagdata->Dim, sizeof(int));
//	//cudaMalloc((void**)&cuda_imagdata->rawSize, sizeof(int));
//	//cudaMalloc((void**)&cuda_imagdata->imgSize, sizeof(int));
//
//	//cudaMalloc((void**)&cuda_imagdata, sizeof(Imagdata));
//	cudaMalloc((void**)&cuda_imagdata->data, sizeof(cBYTE) * imagdata->rawSize);
//
//	printf("2.1\n");
//
//	//cudaMalloc((void**)&cuda_imagecov->row, sizeof(int));
//	//cudaMalloc((void**)&cuda_imagecov->column, sizeof(int));
//
//
//	//cudaMalloc((void**)&cuda_imagecov, sizeof(CovData));
//
//	cuda_imagecov->data = new cBYTE[sizeof(cBYTE) * imagdata->rawSize];
//	cudaMalloc((void**)&cuda_imagecov->data, sizeof(covBYTE) * imagdata->Dim * imagdata->Dim);
//
//	printf("2.3\n");
//
//	cudaMalloc((void**)&cuda_datatemp, sizeof(cBYTE) * imagdata->rawSize);
//
//	cudaMalloc((void**)&time, sizeof(clock_t) * blocks_num * 2);
//
//	printf("3\n");
//
//
//	//cudaMemcpy 将产生的矩阵复制到显卡内存中
//	//cudaMemcpyHostToDevice - 从内存复制到显卡内存
//	//cudaMemcpyDeviceToHost - 从显卡内存复制到内存
//
//	//cudaMemcpy(cuda_imagdata, imagdata, sizeof(cBYTE) * imagdata->rawSize, cudaMemcpyHostToDevice);
//	cudaMemcpy(cuda_imagdata->data, imagdata->data, sizeof(cBYTE) * imagdata->rawSize, cudaMemcpyHostToDevice);
//
//	//cudaMemcpy(cuda_imagecov, imagecov, sizeof(covBYTE) * imagdata->rawSize, cudaMemcpyHostToDevice);
//	cudaMemcpy(cuda_imagecov->data, imagecov->data, sizeof(covBYTE) * imagdata->rawSize, cudaMemcpyHostToDevice);
//
//	cudaMemcpy(cuda_datatemp, datatemp, sizeof(cBYTE) * imagdata->rawSize, cudaMemcpyHostToDevice);
//
//	printf("4\n");
//
//	// 在CUDA 中执行函数 语法：函数名称<<<block 数目, thread 数目, shared memory 大小>>>(参数...);
//	//matMultCUDA <<< blocks_num, THREAD_NUM>>> (cuda_imag;data, cuda_imagecov, cuda_datatemp, time);
//	cudatest << < 1, 64 >> > ;
//
//	cudaError_t cudaStatus = cudaGetLastError();
//	if (cudaStatus != cudaSuccess)
//	{
//		fprintf(stderr, "addKernel launch failed: %s\n", cudaGetErrorString(cudaStatus));
//	}
//
//
//	printf("5\n");
//
//	/*把结果从显示芯片复制回主内存*/
//
//	clock_t time_use[blocks_num * 2];
//
//	//cudaMemcpy 将结果从显存中复制回内存
//	Imagdata* new_imagdata;
//	CovData* new_imagecov;
//
//	new_imagdata = new Imagdata[sizeof(Imagdata)]; // 申请结构体数组（指针）的内存
//	new_imagecov = new CovData[sizeof(CovData)];
//
//	//cudaMemcpy(new_imagdata, cuda_imagdata, sizeof(cBYTE) * imagdata->rawSize, cudaMemcpyDeviceToHost);
//	cudaMemcpy(new_imagdata->data, cuda_imagdata->data, sizeof(cBYTE) * imagdata->rawSize, cudaMemcpyDeviceToHost);
//	//cudaMemcpy(new_imagecov, cuda_imagecov, sizeof(covBYTE) * imagdata->rawSize, cudaMemcpyDeviceToHost);
//	cudaMemcpy(new_imagecov->data, cuda_imagecov->data, sizeof(covBYTE) * imagdata->rawSize, cudaMemcpyDeviceToHost);
//
//	cudaMemcpy(&time_use, time, sizeof(clock_t) * blocks_num * 2, cudaMemcpyDeviceToHost);
//
//	printf("6\n");
//
//	//Free
//	cudaFree(cuda_imagdata->data);
//	//cudaFree(cuda_imagdata);
//
//	cudaFree(cuda_imagecov->data);
//	//cudaFree(cuda_imagdata);
//
//	cudaFree(cuda_datatemp);
//	cudaFree(time);
//
//	//把每个 block 最早的开始时间，和最晚的结束时间相减，取得总运行时间
//	clock_t min_start, max_end;
//
//	min_start = time_use[0];
//
//	max_end = time_use[blocks_num];
//
//	for (int i = 1; i < blocks_num; i++)
//	{
//		if (min_start > time_use[i]) min_start = time_use[i];
//
//		if (max_end < time_use[i + blocks_num]) max_end = time_use[i + blocks_num];
//	}
//
//	//核函数运行时间
//	clock_t final_time = max_end - min_start;
//
//	printf("gputime: %d\n", final_time);
//
//}
